#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "util.hpp"

int main(int argc, char** argv) {
    size_t pow = read_arg(argc, argv, 1, 16);
    size_t n = 1 << pow;
    auto size_in_bytes = n * sizeof(double);

    std::cout << "sort test of length n = " << n
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << std::endl;

    // fill a vector with random values
    thrust::host_vector<double>   values_host(n);
    thrust::device_vector<double> values_device(n);

    // start the nvprof profiling
    hipProfilerStart();

    // copy values to device
    auto start = get_time();
    values_device = values_host;

    // sort values
    thrust::sort(thrust::device, values_device.begin(), values_device.end());

    auto time_taken = get_time() - start;

    std::cout << "time : " << time_taken << "s\n";

    // copy result back to host
    values_host = values_device;

    // check for errors
    bool pass = std::is_sorted(values_host.begin(), values_host.end());

    // stop the profiling session
    hipProfilerStop();

    std::cout << (pass ? "passed\n" : "failed\n");

    return 0;
}

